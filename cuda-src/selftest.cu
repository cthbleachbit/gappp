#include "hip/hip_runtime.h"
#ifndef __global__
#define __global__
#endif

#include "gappp_types.h"
#include "l3fwd.h"
#include "Logging.h"

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <iostream>

#define GAPPP_LOG_SELFTEST "GPU Self Test"

namespace GAPPP {
	namespace selftest {
		static int device_id = -1;

		__global__ void GenVec(double base, double*A) {
			int i = threadIdx.x;
			A[i] = pow(2, base + 2 * i);
		}

		template<typename T>
		struct aligned_vec128 {
			alignas(4096) T v[128];
		};

		// Mapping CPU memory to GPU:
		// https://github.com/NVIDIA/cuda-samples/blob/master/Samples/0_Introduction/simpleZeroCopy/simpleZeroCopy.cu
		int invoke(unsigned int nbr_tasks, struct rte_mbuf **packets) {
			int device_count = 0;
			hipDeviceProp_t device_properties{};

			hipError_t ret = hipGetDeviceCount(&device_count);
			if (device_count < 0) {
				whine(Severity::CRIT, "No CUDA devices found", GAPPP_LOG_SELFTEST);
			} else {
				whine(Severity::INFO, fmt::format("{} CUDA devices found", device_count), GAPPP_LOG_SELFTEST);
			}
			device_id = 0;
			hipSetDevice(device_id);
			ret = hipGetDeviceProperties(&device_properties, device_id);
			if (ret < 0) {
				whine(Severity::CRIT, "Failed to obtain CUDA device properties", GAPPP_LOG_SELFTEST);
			}

			whine(Severity::INFO, fmt::format("Using CUDA device {}", device_properties.name), GAPPP_LOG_SELFTEST);

			ret = hipSetDeviceFlags(hipDeviceMapHost);
			if (ret < 0) {
				whine(Severity::CRIT, "Device does not support mapping from host memory", GAPPP_LOG_SELFTEST);
			}

			auto va = new aligned_vec128<double>(); // 4K aligned allocation
			double *da; // GPU memory pointer - not valid under CPU content

			// Maps va->v into GPU memory space
			ret = hipHostRegister(va->v, 128 * sizeof(double), hipHostRegisterMapped);
			if (ret < 0) {
				whine(Severity::CRIT, "Failed to map from host memory", GAPPP_LOG_SELFTEST);
			}

			// Obtain memory pointers in GPU memory space - this pointer will be passed to CUDA kernel routines
			ret = hipHostGetDevicePointer((void **) &da, (void *)va->v, 0);
			if (ret < 0) {
				whine(Severity::CRIT, "Failed to obtain mapped GPU memory address", GAPPP_LOG_SELFTEST);
			}

			dim3 block(256);
			dim3 grid((unsigned int)ceil(128 / (float)block.x));
			GenVec<<<grid, block>>>(0, da);
			hipDeviceSynchronize();
			ret = hipGetLastError();
			if (ret < 0) {
				whine(Severity::CRIT, "Self test vector generation failed", GAPPP_LOG_SELFTEST);
			}

			// Compare results
			bool results = va->v[0] == 1.0f && va->v[1] == 4.0f;
			delete va;

			return results ? 0 : -1;
		}
	}
}